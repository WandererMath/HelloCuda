
#include <hip/hip_runtime.h>
#include <stdio.h>

// Kernel definition
__global__ void VecAdd(int* A, int* B, int* C)
{
    int i = threadIdx.x;
    C[i] = A[i] + B[i];
    //printf("threadIDx: %d\n", i);
}


void initVec(int* A, int n){
    for(int i=0; i<n;i++){
        A[i]=i;
    }
}

void printVec(int* A, int n){
    for(int i=0; i<n;i++){
        printf("Element %d:  %d\n", i, A[i]);
    }
}

int main()
{
    const int N=100;
    int A[N];
    int B[N];
    int C[N];

    initVec(A, N);
    initVec(B,N);
    // Kernel invocation with N threads
    VecAdd<<<1, N>>>(A, B, C);
    printVec(C, N);

}