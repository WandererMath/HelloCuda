
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 10

// Kernel function to add two vectors
__global__ void vectorAdd(int *a, int *b, int *c) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N) {
        c[tid] = a[tid] + b[tid];
    }
}

int main() {
    int a[N], b[N], c[N];  // Host vectors
    int *d_a, *d_b, *d_c;  // Device vectors

    // Initialize input vectors
    for (int i = 0; i < N; ++i) {
        a[i] = i;
        b[i] = i * 2;
    }

    // Allocate memory on the device
    hipMalloc(&d_a, N * sizeof(int));
    hipMalloc(&d_b, N * sizeof(int));
    hipMalloc(&d_c, N * sizeof(int));

    // Copy input vectors from host to device memory
    hipMemcpy(d_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel on the GPU
    vectorAdd<<<1, N>>>(d_a, d_b, d_c);

    // Copy result vector from device to host memory
    hipMemcpy(c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Print the result
    for (int i = 0; i < N; ++i) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    return 0;
}
